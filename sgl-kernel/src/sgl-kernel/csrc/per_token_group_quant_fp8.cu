#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Float8_e4m3fn.h>

#include <cmath>

#ifndef USE_ROCM
#include <flashinfer/vec_dtypes.cuh>
#else
#include "hip_vec_dtypes.h"
#endif

#include "utils.h"

#ifndef USE_ROCM
#define FULL_MASK 0xffff
#else
#define FULL_MASK 0xffffffff
#endif

#define kGROUPS_PER_BLOCK   16
#define kTHRS_PER_GROUP     16
#define kBitsToLoad         128
#define kBytesToLoad        (kBitsToLoad / 8)

#ifndef USE_ROCM
using FP8_TYPE = c10::Float8_e4m3fn;
#else
using FP8_TYPE = c10::Float8_e4m3fnuz;
#endif

__device__ __forceinline__ float GroupReduce(float val) {
  val = fmaxf(val, __shfl_xor_sync(FULL_MASK, val, 8));
  val = fmaxf(val, __shfl_xor_sync(FULL_MASK, val, 4));
  val = fmaxf(val, __shfl_xor_sync(FULL_MASK, val, 2));
  val = fmaxf(val, __shfl_xor_sync(FULL_MASK, val, 1));
  return val;
}

template <typename T>
__global__ void per_token_group_quant_fp8_kernel(const T* __restrict__ input, void* __restrict__ output_q,
                                                 float* __restrict__ output_s, const int group_size,
                                                 const int num_groups, const float eps, const float fp8_min,
                                                 const float fp8_max) {
  const int groups_per_block = kGROUPS_PER_BLOCK;
  const int local_group_id = threadIdx.x / kTHRS_PER_GROUP;
  const int lane_id = threadIdx.x % kTHRS_PER_GROUP;

  const int block_group_id = blockIdx.x * groups_per_block;
  const int block_group_offset = (block_group_id + local_group_id) * group_size;

  __shared__ float s_absmax[kGROUPS_PER_BLOCK];

  float local_absmax = eps;

  const T* group_input = input + block_group_offset;
  FP8_TYPE* group_output = static_cast<FP8_TYPE*>(output_q) + block_group_offset;
  float* scale_output = output_s + block_group_id + local_group_id;

  constexpr uint32_t vec_size = kBytesToLoad / sizeof(T);
  using vec_t = flashinfer::vec_t<T, vec_size>;

  const int32_t num_vec_elems = group_size / vec_size;

  for (int32_t i = lane_id; i < num_vec_elems; i += kTHRS_PER_GROUP) {
    vec_t input_vec;
    input_vec.cast_load(group_input + i * vec_size);

#pragma unroll
    for (uint32_t j = 0; j < vec_size; ++j) {
      float val = castToFloat(input_vec[j]);
      float abs_val = fabsf(val);
      local_absmax = fmaxf(local_absmax, abs_val);
    }
  }

  local_absmax = GroupReduce(local_absmax);

  if (lane_id == 0) {
    s_absmax[local_group_id] = local_absmax;
  }
  __syncthreads();

  const float group_absmax = s_absmax[local_group_id];
  const float y_s = group_absmax / fp8_max;

  if (lane_id == 0) {
    *scale_output = y_s;
  }

  for (int32_t i = lane_id; i < num_vec_elems; i += kTHRS_PER_GROUP) {
    vec_t input_vec;
    input_vec.cast_load(group_input + i * vec_size);

#pragma unroll
    for (uint32_t j = 0; j < vec_size; ++j) {
      float val = castToFloat(input_vec[j]);
      float q_val = fminf(fmaxf(val / y_s, fp8_min), fp8_max);
      group_output[i * vec_size + j] = FP8_TYPE(q_val);
    }
  }
}

void sgl_per_token_group_quant_fp8(torch::Tensor input, torch::Tensor output_q, torch::Tensor output_s,
                                   int64_t group_size, double eps, double fp8_min, double fp8_max) {
  CHECK_INPUT(input);
  CHECK_INPUT(output_q);
  CHECK_INPUT(output_s);

  const int num_groups = input.numel() / group_size;

  CHECK_EQ(input.numel() % group_size, 0);

  dim3 grid(CEILDIV(num_groups, kGROUPS_PER_BLOCK));
  dim3 block(kGROUPS_PER_BLOCK * kTHRS_PER_GROUP);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {
    per_token_group_quant_fp8_kernel<scalar_t><<<grid, block, 0, stream>>>(
        static_cast<scalar_t*>(input.data_ptr()), output_q.data_ptr(), static_cast<float*>(output_s.data_ptr()),
        group_size, num_groups, (float)eps, (float)fp8_min, (float)fp8_max);
    return true;
  });
}
